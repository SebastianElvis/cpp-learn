/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16;

// name             executed by           invoked by
// __device__        device                 device
// __global__        device                 host
// __host__          host                   host


// Every thread executes this kernel function simultaneously
// so called SIMT
__global__
void vector_add(int *a, int *b, int *res)
{
	// blockIdx: block id
	// blockDim: threads per block
	// threadIdx: thread id
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	res[i] = a[i] + b[i];
}

void init(int x[])
{
	for(int i=0;i<N;i++){
		x[i] = i;
	}
}

int main()
{
	int a[N];
	int b[N];
	int res[N];

	init(a);
	init(b);

	int *ad, *bd, *resd;
	int copy_size = N*sizeof(int);

	hipMalloc(&ad, copy_size);
	hipMalloc(&bd, copy_size);
	hipMalloc(&resd, copy_size);
	hipMemcpy(ad, a, copy_size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, copy_size, hipMemcpyHostToDevice);

	// kernel_func<<<numBlock, numThread, Ns, S>>>()
	// <<<numBlock, numThread, Ns, S>>>
	// Ns, S optional
	// Ns: the maximum dynamic memory size
	// the kernel function is in the stream S


	//vector_add<<<N, 1>>>(ad, bd, resd); // Totally 1xN=N threads, it works!
	//vector_add<<<1, N>>>(ad, bd, resd); // Totally 1xN=N threads, it works!
	//vector_add<<<1, (N-1)>>>(ad, bd, resd); // Totally (N-1) threads, so omits the last element...
	vector_add<<<4, 4>>>(ad, bd, resd); // Totally 4x4=N(16) threads, it works!
	hipMemcpy(res, resd, copy_size, hipMemcpyDeviceToHost);

	hipFree(ad);
	hipFree(bd);
	hipFree(resd);

	for(int i=0;i<N;i++){
		printf("%d ", res[i]);
	}
	return EXIT_SUCCESS;
}
