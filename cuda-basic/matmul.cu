/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 16

void init_mat(int mat[N][N]){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			mat[i][j] = rand() % 100;
		}
	}
}

void init_i(int mat[N][N]){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			if(i == j) mat[i][j] = 1;
			else mat[i][j] = 0;
		}
	}
}

void init_zeros(int mat[N][N]){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			mat[i][j] = 0;
		}
	}
}

void print_mat(int mat[N][N]){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			printf("%3d ", mat[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void print_mat2(int *mat){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			printf("%3d ", mat[i*N+j]);
		}
		printf("\n");
	}
	printf("\n");
}

__global__
void mat_mul(int *x, int *y, int *z){
	int id = blockIdx.x *blockDim.x + threadIdx.x;
	int row = id/N;
	int col = id%N;
	z[row*N+col] = 0;
	for(int i=0;i<N;i++){
		z[row*N+col] += x[row*N+i] * y[i*N+col];
	}
}

int main(void) {
	int x[N][N], y[N][N], z[N][N];

	// cannot be prined
	// device functions cannot invoke host functions
	// otherwise add __device__ before the function you want to invoke
	int *xd, *yd, *zd;

	int mat_size = N*N*sizeof(int);

	init_mat(x);
	init_i(y);
	init_zeros(z);

	print_mat(x);

	hipMalloc(&xd, mat_size);
	hipMalloc(&yd, mat_size);
	hipMalloc(&zd, mat_size);

	hipMemcpy(xd, x, mat_size, hipMemcpyHostToDevice);
	hipMemcpy(yd, y, mat_size, hipMemcpyHostToDevice);
	hipMemcpy(zd, z, mat_size, hipMemcpyHostToDevice);


	mat_mul<<<N, N>>>(xd, yd, zd);

	hipMemcpy(z, zd, mat_size, hipMemcpyDeviceToHost);

	print_mat(z);
}
