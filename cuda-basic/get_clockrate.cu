#include <stdio.h>



int get_GPU_Rate()
{
 hipDeviceProp_t deviceProp;
 hipGetDeviceProperties(&deviceProp,0);
 return deviceProp.clockRate;
}

int main() {
	printf("GPU Rate is %d\n", get_GPU_Rate());
}
